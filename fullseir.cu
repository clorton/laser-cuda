#include "hip/hip_runtime.h"
// Helper for XOROSHIRO128+ PRNG
__device__ inline uint64_t rotl(uint64_t x, uint32_t k) {
    return (x << k) | (x >> (64 - k));
}

__device__ uint64_t get_next(uint64_t *states, uint32_t index) {

    uint64_t s0 = states[index*2];
    uint64_t s1 = states[index*2+1];
    uint64_t result = s0 + s1;

    s1 ^= s0;
    s0 = rotl(s0, 55) ^ s1 ^ (s1 << 14);
    s1 = rotl(s1, 36);

    states[index*2] = s0;
    states[index*2+1] = s1;

    return result;
}

#define FACTOR (double(1.0) / 9007199254740992)

__device__ inline float uint64_to_unit_float32(uint64_t x) {
    return float((x >> 11) * FACTOR);
}

#define TWO_PI  (float(2.0 * 3.14159265))

__device__ float draw_normal(uint64_t s0, uint64_t s1, float mean, float std) {

    float u1 = uint64_to_unit_float32(s0);
    float u2 = uint64_to_unit_float32(s1);
    float z0 = sqrt(-2 * log(u1)) * cos(TWO_PI * u2);
    return (z0 * std + mean);
}

__global__ void infect(uint32_t count, uint32_t *indices, uint64_t *prng_states, uint8_t *susceptibility, uint8_t *itimer, float inf_mean, float inf_std) {
    int ithread = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ uint64_t prns[];
    if (ithread < count) {
        if (ithread == 0) {
            uint64_t *pprns = prns;
            for (int j = 0; j < count; ++j) {
                *pprns++ = get_next(prng_states, 0);
                *pprns++ = get_next(prng_states, 0);
            }
        }
        __syncthreads();    // Wait here until all PRNS have been generated.
        uint32_t iagent = indices[ithread];
        uint32_t iS0 = ithread * 2;
        uint32_t iS1 = iS0 + 1;
        susceptibility[iagent] = 0;
        itimer[iagent] = uint8_t(__roundf(draw_normal(prns[iS0], prns[iS1], inf_mean, inf_std)));
    }
}

__global__ void infection_update(uint32_t count, uint8_t *itimer) {
    int iagent = blockIdx.x * blockDim.x + threadIdx.x;
    if (iagent < count) {
        if (itimer[iagent] > 0) {
            itimer[iagent] -= 1;
        }
    }
}

__global__ void exposure_update(
    uint32_t count,
    uint8_t *etimer,
    uint64_t *prng_states,
    uint32_t stride,
    uint8_t *itimer,
    float inf_mean,
    float inf_std)
    {
    int iagent = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ uint64_t prns[];
    if (iagent < count) {
        /*
        Use the initial PRN element to store the infection status.
        Assume no one is getting infected.
         */
        uint32_t generatorS0 = (threadIdx.x - threadIdx.x % stride) * 2;

        bool isGenerator = (threadIdx.x % stride == 0);
        if (isGenerator) {
            prns[generatorS0] = 0;  // Each thread gets two PRNs.
        }
        __syncthreads();            // Wait here until PRN[iprng*2] is set to 0 by the prng thread.

        /*
        Decrement the exposure counter, if non-zero.
        If it hits zero, note in PRN[0] that at least one agent is now infected.
        Remember this in `infected`.
        */
        bool infectious = false;
        if (etimer[iagent] > 0) {
            uint8_t newtimer = etimer[iagent] - 1;
            etimer[iagent] = newtimer;
            if (newtimer == 0) {
                prns[generatorS0] = 1;  // Each thread gets two PRNs.
                infectious = true;
            }
        }
        __syncthreads();    // Wait here until everyone has a chance to set PRN[iprng*2].

        if (isGenerator && (prns[generatorS0] != 1)) {
            /*
            Each thread in `stride` gets two PRNs in case it needs to do a normal
            distribution draw for the duration of infectiousness.
            */
            uint64_t *pprns = prns + generatorS0;
            uint32_t istate = iagent / stride;  // E.g. 0, 0, 0, 0, 1, 1, 1, 1, ... based on `stride`
            for (int j = 0; j < stride; ++j) {
                *pprns++ = get_next(prng_states, istate);
                *pprns++ = get_next(prng_states, istate);
            }
        }
        __syncthreads();    // Wait here until all PRNS have been generated.

        if (infectious) {
            uint32_t myprnS0 = threadIdx.x * 2;
            uint32_t myprnS1 = myprnS0 + 1;
            itimer[iagent] = uint8_t(__roundf(draw_normal(prns[myprnS0], prns[myprnS1], inf_mean, inf_std)));
        }
    }
}

__global__ void transmission_update(
    uint32_t count,
    uint64_t *prng_states,
    uint32_t stride,
    float force,
    uint8_t *susceptibility,
    uint8_t *etimer,
    float exp_mean,
    float exp_std)
    {
    int iagent = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ uint64_t prns[];
    if (iagent < count) {
        uint32_t generatorS0 = (threadIdx.x - threadIdx.x % stride) * 2;
        uint32_t generatorS1 = generatorS0 + 1;
        bool isGenerator = (threadIdx.x % stride == 0);
        if (isGenerator) {
            uint32_t istate = iagent / stride;
            uint64_t *pprns = prns + generatorS0;
            for (int j = 0; j < stride; ++j) {
                *pprns = get_next(prng_states, istate);
                pprns += 2;
            }
            prns[generatorS1] = 0;  // Each thread gets two PRNs.
        }
        __syncthreads();    // Wait here until all PRNS have been generated.

        bool exposed = false;
        uint32_t myprnS0 = threadIdx.x * 2;
        float uniform_draw = uint64_to_unit_float32(prns[myprnS0]);
        if (uniform_draw < (force * susceptibility[iagent])) {
            susceptibility[iagent] = 0;
            prns[generatorS1] = 1;
            exposed = true;
        }
        __syncthreads();    // Wait here until everyone has a chance to set PRNS[generatorS1].

        if (isGenerator && (prns[generatorS1] != 0)) {
            /*
            Each thread in `stride` gets two PRNs in case it needs to do a normal
            distribution draw for the duration of exposure.
            */
            uint32_t istate = iagent / stride;
            uint64_t *pprns = prns + generatorS0;
            for (int j = 0; j < stride; ++j) {
                *pprns++ = get_next(prng_states, istate);
                *pprns++ = get_next(prng_states, istate);
            }
        }
        __syncthreads();    // Wait here until all PRNS have been generated.

        if (exposed) {
            uint32_t myprnS1 = myprnS0 + 1;
            etimer[iagent] = uint8_t(round(draw_normal(prns[myprnS0], prns[myprnS1], exp_mean, exp_std)));
        }
    }
}